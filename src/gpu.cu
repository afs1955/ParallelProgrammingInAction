#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>

#include "timer.h"

using namespace std;

template <typename T>
struct Mat {
  Mat(int width, int height) : _width(width), _height(height), _data(nullptr) {}
  Mat(int width, int height, T init_value)
      : _width(width), _height(height), _data(nullptr) {
    _data = (T *)malloc(sizeof(T) * _width * _height);
    for (int i = 0; i < _width * _height; i++) _data[i] = init_value;
  }
  ~Mat() {
    if (_data != nullptr) free(_data);
  }
  __host__ __device__ T get(int x, int y) { return _data[y * _width + x]; }
  __device__ void set(int x, int y, T value) { _data[y * _width + x] = value; }

  int _height;
  int _width;
  T *_data;
};

__global__ void blur_mat(Mat<float> *input, Mat<float> *output) {
  int width = input->_width;
  int height = input->_height;

  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int left = x - 1 < 0 ? 0 : x - 1;
  int right = x + 1 >= width ? width - 1 : x + 1;
  int above = y - 1 < 0 ? 0 : y - 1;
  int below = y + 1 >= height ? height - 1 : y + 1;

  float res = (input->get(x, y) + input->get(left, y) + input->get(right, y) +
               input->get(x, above) + input->get(left, above) +
               input->get(right, above) + input->get(x, below) +
               input->get(left, below) + input->get(right, below)) /
              9;
  output->set(x, y, res);
}

__global__ void blur_mat_redup(Mat<float> *input, Mat<float> *output) {
  int width = input->_width;
  int height = input->_height;

  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int left = x - 1 < 0 ? 0 : x - 1;
  int right = x + 1 >= width ? width - 1 : x + 1;
  int above = y - 1 < 0 ? 0 : y - 1;
  int below = y + 1 >= height ? height - 1 : y + 1;

  output->set(
      x, y,
      (input->get(x, y) + input->get(left, y) + input->get(right, y)) / 3);
  __syncthreads();
  output->set(
      x, y,
      (output->get(x, y) + output->get(x, above) + output->get(x, below)) / 3);
}

void print_mat(Mat<float> &mat) {
  for (int y = 0; y < mat._height; y++) {
    for (int x = 0; x < mat._width; x++) {
      cout << mat.get(x, y) << ", ";
    }
    cout << endl;
  }
}

int main() {
  hipSetDevice(3);

  const int width = 8192;
  const int height = 4096;

  Mat<float> *input = new Mat<float>(width, height, 0.0f);
  Mat<float> *output = new Mat<float>(width, height, 0.0f);

  Mat<float> *d_input;
  Mat<float> *d_output;
  Mat<float> *d_input_data = new Mat<float>(width, height);
  Mat<float> *d_output_data = new Mat<float>(width, height);

  hipMalloc((void **)&d_input, sizeof(Mat<float>));
  hipMalloc((void **)&d_output, sizeof(Mat<float>));
  hipMalloc((void **)&(d_input_data->_data), sizeof(float) * width * height);
  hipMalloc((void **)&(d_output_data->_data), sizeof(float) * width * height);
  hipMemcpy(d_input, d_input_data, sizeof(Mat<float>), hipMemcpyHostToDevice);
  hipMemcpy(d_output, d_output_data, sizeof(Mat<float>),
             hipMemcpyHostToDevice);

  Timer t_copy("Host to device");
  hipMemcpy(d_input_data->_data, input->_data, sizeof(float) * width * height,
             hipMemcpyHostToDevice);
  t_copy.stop();

  dim3 dim_block(16, 8);
  dim3 dim_grid(width / dim_block.x, height / dim_block.y);

  Timer t1("original");
  blur_mat<<<dim_grid, dim_block>>>(d_input, d_output);
  hipDeviceSynchronize();
  t1.stop();

  hipMemcpy(output->_data, d_output_data->_data,
             sizeof(float) * width * height, hipMemcpyDeviceToHost);

  Timer t2("redup");
  blur_mat_redup<<<dim_grid, dim_block>>>(d_input, d_output);
  hipDeviceSynchronize();
  t2.stop();

  hipMemcpy(output->_data, d_output_data->_data,
             sizeof(float) * width * height, hipMemcpyDeviceToHost);

  printf("%f %f %f\n", output->get(0, 0), output->get(1, 0),
         output->get(100, 100));

  return 0;
}
